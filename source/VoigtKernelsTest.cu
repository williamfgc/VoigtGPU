#include "hip/hip_runtime.h"
/*
 * VoigtKernelsTest.cpp
 *
 *  Created on: May 10, 2017
 *      Author: wgodoy
 */
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "VoigtFunctions.h"

int main()
{

    const unsigned int size = 10;

    //    thrust::host_vector<float> bLW(size, 1.f);
    //    thrust::device_vector<float> bandLineWidths = bLW;

    thrust::device_vector<float> bandLineWidths(size, 1.f);
    //    thrust::device_vector<float> temperatureReference(1, 1.f);
    //    thrust::device_vector<float> temperatures(size, 1.f);
    //    thrust::device_vector<float> wavenumberReference(1, 1.f);
    //    thrust::device_vector<float> wavenumbers(size, 1.f);
    //    thrust::device_vector<float2> voigtXY(size);

    for (unsigned int i = 0; i < size; i++)
    {
        std::cout << "bandLineWidths[" << i << "] = " << bandLineWidths[i]
                  << "\n";
    }

    //    voigt::SetComplexInputs(bandLineWidths, temperatureReference,
    //    temperatures,
    //                            wavenumberReference, wavenumbers, voigtXY);

    return 0;
}
